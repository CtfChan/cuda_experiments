
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_BLOCKS 1
#define BLOCK_WIDTH 256

__global__ void hello()
{
	printf("Hello world! I'm thread %d\n", threadIdx.x );
}

int main(int argc, char **argv) 
{
	hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

	hipDeviceSynchronize();

	printf("That is all!\n");

	return 0;
}