
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void)
{
	printf("Hello World from GPU\n");
}

int main(void)
{
	printf("Hello world from CPU \n");

	helloFromGPU<<<1, 10>>>();
	hipDeviceReset();
	return 0;
}




// nvcc -arch sm_20 hello.cu -o hello
