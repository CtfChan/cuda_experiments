
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void)
{
	printf("Hello World from GPU\n");
}

int main(void)
{
	printf("Hello world from CPU \n");

	helloFromGPU<<<1, 10>>>();
	hipDeviceReset();
	return 0;
}




